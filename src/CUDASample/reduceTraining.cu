#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

typedef float(*reduceFPtr)(float,float);

__device__ float d_Max(float a, float b)
{
    return max(a, b);
}

__device__ float d_Add(float a, float b)
{
    return a+b;
}

__global__ void reduceFun(float *input, unsigned int s, float *output, reduceFPtr func, bool last)
{
    const int tId = blockIdx.x*blockDim.x + threadIdx.x;
    if (tId >= s) return;
    if (!last && tId < s)
    {
        input[tId] = (*func)(input[tId], input[tId+s]);
    }
    if (last) input[s] = input[s+s];
    if (tId == 0) *output = input[0];
}

void invokeReduceFun(float *input, int inputSize, float *output, reduceFPtr func)
{
    dim3 blockSize(1,1,1);
    dim3 gridSize(1,1,1);
    bool odd = false;
    unsigned int s = inputSize;
    while (s > 1)
    {
        odd = (s%2);
        s >>= 1;
        blockSize.x = 512;
        gridSize.x = (s + blockSize.x - 1) / blockSize.x;
        reduceFun<<<gridSize, blockSize>>>(input, s, output, func, false);
        hipDeviceSynchronize();
        if (odd)
        {
            reduceFun<<<1, 1>>>(input, s, output, func, true);
            hipDeviceSynchronize();
            s++;
        }
    }

}

__device__ reduceFPtr d_p1 = d_Add;

void reduceSample()
{
    printf("Starting reduce sample ...\n");
    dim3 blockSize(1,1,1);
    dim3 gridSize(1,1,1);
    unsigned int tableSize = 1000;
    float *d_tableToReduce;
    float *h_tableToReduce;
    float *d_result;
    float *h_result;
    reduceFPtr h_p1;
    checkCudaErrors(hipMemcpyFromSymbol(&h_p1, HIP_SYMBOL(d_p1), sizeof(reduceFPtr)));

    h_tableToReduce = (float*) malloc(sizeof(float)*tableSize);
    h_result = (float*) malloc(sizeof(float));
    checkCudaErrors(hipMalloc(&d_tableToReduce, sizeof(float)*tableSize));
    checkCudaErrors(hipMalloc(&d_result, sizeof(float)));

    for (unsigned int i=0; i<tableSize; i++) h_tableToReduce[i]=(float)i;

    checkCudaErrors(hipMemcpy(d_tableToReduce, h_tableToReduce, sizeof(float)*tableSize, hipMemcpyHostToDevice));
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    invokeReduceFun(d_tableToReduce, tableSize, d_result, h_p1);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(h_result, d_result, sizeof(float), hipMemcpyDeviceToHost));

    printf("Obtained result: %f\n",*h_result);
    checkCudaErrors(hipFree(d_tableToReduce));
    checkCudaErrors(hipFree(d_result));
    free(h_tableToReduce);
    free(h_result);
}


